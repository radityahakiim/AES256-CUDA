#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "aes_header.cuh"
#include <iostream>
#include <vector>

__global__ void AddRoundKey(uint8_t* state, const uint8_t* roundKey) {
	int idx = threadIdx.x;
	if (idx < Nb * 4) {
		state[idx] ^= roundKey[idx];
	}
}