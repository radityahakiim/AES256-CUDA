#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "aes_header.cuh"
#include <iostream>
#include <vector>

__device__ uint8_t galois_mul2(uint8_t x) {
    return (x << 1) ^ ((x & 0x80) ? 0x1B : 0x00);
}

__device__ uint8_t galois_mul3(uint8_t x) {
    return galois_mul2(x) ^ x;
}

__global__ void MixColumnsKernel(uint8_t* state) {
    int col = threadIdx.x;

    if (col < 4) {
        uint8_t s0 = state[col];
        uint8_t s1 = state[4 + col];
        uint8_t s2 = state[8 + col];
        uint8_t s3 = state[12 + col];

        state[col]      = galois_mul2(s0) ^ galois_mul3(s1) ^ s2 ^ s3;
        state[4 + col]  = s0 ^ galois_mul2(s1) ^ galois_mul3(s2) ^ s3;
        state[8 + col]  = s0^s1 ^ galois_mul2(s2) ^ galois_mul3(s3); 
        state[12 + col] = galois_mul3(s0) ^ s1 ^ s2 ^ galois_mul2(s3);
    }
}