#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "aes_header.cuh"
#include <iostream>
#include <vector>

__global__ void SubBytesKernel(uint8_t* state) {
	int idx = threadIdx.x;

	if (idx < AES_BLOCK_SIZE) {
		state[idx] = sbox[state[idx]];
	}
}