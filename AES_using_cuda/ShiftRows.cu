#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "aes_header.cuh"
#include <iostream>
#include <vector>

__global__ void ShiftRowsKernel(uint8_t* state) {
	int idx = threadIdx.x;

	if (idx == 1) {
		// Second row shifts one position to left
		uint8_t temp = state[1];
		state[1]     = state[5];
		state[5]     = state[9];
		state[9]     = state[13];
		state[13]    = temp;
	}
	else if (idx == 2) {
		// Third row shift two positions to the left
		uint8_t temp1 = state[2];
		uint8_t temp2 = state[6];
		state[2]      = state[10];
		state[6]      = state[14];
		state[10]     = temp1;
		state[14]     = temp2;
	}
	else if (idx == 3) {
		uint8_t temp = state[3];
		state[3]     = state[15];
		state[15]    = state[11];
		state[11]    = state[7];
		state[7]     = temp;
	}
}